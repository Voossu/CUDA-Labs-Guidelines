#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define SIZE 512

// ������� ��������� ���� �������
__global__ void addVector(float* left, float* right, float* result) {
	// �������� id ������� �����.
	int idx = blockIdx.x;
	// ����������� ���������.
	result[idx] = left[idx] + right[idx];
}

__host__ int main() {

	// �������� ���'��� �� �������
	float *vec1 = new float[SIZE], *vec2 = new float[SIZE], *vec3 = new float[SIZE];
	// ����������� �������� �������
	for (int i = 0; i < SIZE; i++) vec1[i] = vec2[i] = i;

	// ��������� �� ���'��� ���������
	float *devVec1, *devVec2, *devVec3;
	// �������� ���'��� ��� ������� �� ��������
	hipMalloc((void**)&devVec1, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec2, sizeof(float) * SIZE);
	hipMalloc((void**)&devVec3, sizeof(float) * SIZE);
	// ������� ��� � ���'��� ���������
	hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice);

	// ����� ����, �� ���������������
	dim3 gridSize = dim3(SIZE, 1, 1);
	// ����� �����, �� ���������������
	dim3 blockSize = dim3(1, 1, 1);
	// �������� ������ ������� ����
	addVector << <gridSize, blockSize >> >(devVec1, devVec2, devVec3);

	// �������� ��䳿
	hipEvent_t syncEvent;
	// ��������� ����
	hipEventCreate(&syncEvent);
	// �������� ����
	hipEventRecord(syncEvent, 0);
	// ����������� ����
	hipEventSynchronize(syncEvent);
	// ҳ���� ����� �������� ��������� ����������
	hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

	// ���������� ����������
	for (int i = 0; i < SIZE; i++) printf("Element #%i: %.1f\n", i, vec3[i]);
	// ������������ �������
	// ��������� ��䳿
	hipEventDestroy(syncEvent);
	// ���������� ���'�� �� ��������
	hipFree(devVec1);
	hipFree(devVec2);
	hipFree(devVec3);
	// ���������� ������ ������� ��������
	delete[] vec1, vec2, vec3;

}