#include "hip/hip_runtime.h"
#include "stdio.h"

int main(int argc, char* argv[]) {

	int deviceCount;
	hipDeviceProp_t deviceProp;

	hipGetDeviceCount(&deviceCount);

	printf("Device count: %d\n\n", deviceCount);

	for (int i = 0; i < deviceCount; i++) {
		hipGetDeviceProperties(&deviceProp, i);

		printf("Device %d name: %s\n", i + 1, deviceProp.name);
		printf("Total global memory: %zu\n", deviceProp.totalGlobalMem);
		printf("Shared memory per block: %zu\n", deviceProp.sharedMemPerBlock);
		printf("Registers per block: %d\n", deviceProp.regsPerBlock);
		printf("Warp size: %d\n", deviceProp.warpSize);
		printf("Memory pitch: %zu\n", deviceProp.memPitch);
		printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
		printf("Max threads dimensions: x = %d, y = %d, z = %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("Max grid size: x = %d, y = %d, z = %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("Clock rate: %d\n", deviceProp.clockRate);
		printf("Total constant memory: %zu\n", deviceProp.totalConstMem);
		printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("Texture alignment: %zu\n", deviceProp.textureAlignment);
		printf("Device overlap: %d\n", deviceProp.deviceOverlap);
		printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
		printf("Kernel execution timeout enabled: %s\n\n", deviceProp.kernelExecTimeoutEnabled ? "true" : "false");
	}

	return 0;
}
